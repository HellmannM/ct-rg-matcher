#include "hip/hip_runtime.h"
// This file is distributed under the MIT license.
// See the LICENSE file for details.

#include "render.h"

namespace visionaray
{

void render_cu(
        cuda_volume_ref_t const&        volume,
        aabb                            bbox,
        host_device_rt&                 rt,
        cuda_sched<ray_type_gpu>&       sched,
        camera_t const&                 cam,
        float                           delta,
        float                           photon_energy
        )
{
    auto sparams = make_sched_params(
            cam,
            rt
            );

    using R = ray_type_gpu;
    using S = R::scalar_type;
    using C = vector<4, S>;

    sched.frame([=] __device__ (R ray, int x, int y) -> result_record<S>
    {
        result_record<S> result;

        //bool debug = (x == 256) && (y == 256);
        //bool crosshair = (x == 256) || (y == 256);
        //if (crosshair) {result.color = C(1.f, 1.f, 1.f, 1.f); result.hit = true; return result;}

        auto hit_rec = intersect(ray, bbox);
        auto t = max(S(0.0), hit_rec.tnear);

        result.color = C(0.0);
        S accumulated_LAC = 0.0;
        size_t steps = 0;

        while ( any(t < hit_rec.tfar) )
        {
            auto pos = ray.ori + ray.dir * t;
            auto tex_coord = vector<3, S>(
                    ( pos.x + (bbox.size().x / 2) ) / bbox.size().x,
                    (-pos.y + (bbox.size().y / 2) ) / bbox.size().y,
                    (-pos.z + (bbox.size().z / 2) ) / bbox.size().z
                    );

            // sample volume
            auto voxel = tex3D(volume, tex_coord);
            accumulated_LAC += select(
                    t < hit_rec.tfar,
                    voxel,
                    0.f);

            // step on
            t += delta;
            ++steps;
        }

        auto average_LAC = accumulated_LAC / steps;
        auto traveled_distance_cm = (steps * delta) / S(10.0); // delta is in [mm]/[px]
        auto fraction_remaining = pow(photon_energy, -traveled_distance_cm * average_LAC);
        result.color = C(1.f) - C(fraction_remaining);

        result.hit = hit_rec.hit;
        return result;
    }, sparams);
}

} // visionaray
