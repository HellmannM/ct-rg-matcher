#include "hip/hip_runtime.h"
// This file is distributed under the MIT license.
// See the LICENSE file for details.

#include "render.h"

namespace visionaray
{

void render_cu(
        cuda_volume_ref_t const&        volume,
        cuda_transfunc_ref_t const&     transfunc,
        aabb                            bbox,
        host_device_rt&                 rt,
        cuda_sched<ray_type_gpu>&       sched,
        camera_t const&                 cam,
        projection_algo                 algo,
        float                           delta
        )
{
    auto sparams = make_sched_params(
            cam,
            rt
            );

    using R = ray_type_gpu;
    using S = R::scalar_type;
    using C = vector<4, S>;

    sched.frame([=] __device__ (R ray, int x, int y) -> result_record<S>
    {
        result_record<S> result;

        bool debug = (x == 280) && (y == 200);
        //bool debug = (x == 256) && (y == 256);
        bool crosshair = (x == 256) || (y == 256);
        //if (debug) printf(".");
        if (crosshair) {result.color = C(1.f, 1.f, 1.f, 1.f); result.hit = true; return result;}

        auto hit_rec = intersect(ray, bbox);
        auto t = hit_rec.tnear;

        result.color = C(0.0);

        while ( any(t < hit_rec.tfar) )
        {
            auto pos = ray.ori + ray.dir * t;
            auto tex_coord = vector<3, S>(
                    ( pos.x + 1.0f ) / 2.0f,
                    (-pos.y + 1.0f ) / 2.0f,
                    (-pos.z + 1.0f ) / 2.0f
                    );

            // sample volume and do post-classification
            auto voxel = tex3D(volume, tex_coord);
            C color = tex1D(transfunc, voxel);

            // premultiplied alpha
            color.xyz() *= color.w;

            // compositing
            if (algo == projection_algo::AlphaCompositing)
            {
                result.color += select(
                        t < hit_rec.tfar,
                        color * (1.0f - result.color.w),
                        C(0.0)
                        );

                // early-ray termination - don't traverse w/o a contribution
                if ( all(result.color.w >= 0.999f) )
                {
                    break;
                }
            }
            else if (algo == projection_algo::MaxIntensity)
            {
                result.color = select(
                        t < hit_rec.tfar,
                        max(color, result.color),
                        result.color
                        );
            }
            else if (algo == projection_algo::MinIntensity)
            {
                result.color = select(
                        t < hit_rec.tfar,
                        min(color, result.color),
                        result.color
                        );
            }
            else if (algo == projection_algo::DRR)
            {
                result.color += select(
                        t < hit_rec.tfar,
                        color,
                        C(0.0)
                        );
            }

            // step on
            t += delta;
        }

        result.hit = hit_rec.hit;
        return result;
    }, sparams);
}

} // visionaray
